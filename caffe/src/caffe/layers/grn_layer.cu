#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/grn_layer.hpp"

namespace caffe {

template <typename Dtype>
void GRNLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  const int num = bottom[0]->num();
  const int channels = bottom[0]->channels();

  caffe_gpu_powx(
      count,
      bottom_data,  // x_i
      Dtype(2),
      bottom_sq_.mutable_gpu_data());  // x_i^2
  caffe_gpu_gemv(
      CblasNoTrans,
      num,
      channels,
      Dtype(1.0),
      bottom_sq_.gpu_data(),  // x_i^2
      summer_vec_.gpu_data(),
      Dtype(0.0),
      norm_sq_.mutable_gpu_data());  // \Sum x_i^2
  caffe_gpu_powx(num, norm_sq_.gpu_data(), Dtype(0.5), norm_.mutable_gpu_data()); // sqrt(\Sum x_i^2)
  caffe_gpu_gemm(
      CblasNoTrans,
      CblasTrans,
      num,
      channels,
      1,
      Dtype(1.0),
      norm_.gpu_data(),  // sqrt(\Sum x_i^2)
      summer_vec_.gpu_data(),
      Dtype(0.0),
      norm_scale_.mutable_gpu_data());
      
  caffe_gpu_div(count, bottom_data, norm_scale_.gpu_data(), top_data);
}

template <typename Dtype>
void GRNLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* top_data = top[0]->gpu_data();
    const int count = bottom[0]->count();
    const int num = bottom[0]->num();
    const int channels = bottom[0]->channels();
    
    caffe_gpu_powx(
      count,
      top_data,  // y_i
      Dtype(2),
      top_sq_.mutable_gpu_data());  // y_i^2
    caffe_gpu_sub(
      count,
      ones_.gpu_data(),  // 1
      top_sq_.gpu_data(),  // y_i^2
      temp_.mutable_gpu_data()); // 1 - y_i^2

    caffe_gpu_div(count, top_diff, norm_scale_.gpu_data(), bottom_diff);

    caffe_gpu_mul(count, temp_.gpu_data(), bottom_diff, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(GRNLayer);

}  // namespace caffe
